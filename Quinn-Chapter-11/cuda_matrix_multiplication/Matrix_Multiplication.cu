#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

// MatrixMul_CUDA.cu
#include <iostream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

using namespace std;

#define TILE_WIDTH 16  // Tile size for shared memory

/*
* Print matrix on screen.
*/
void printMatrix(char* text, const float*& matrix, int rows, int cols) {
    cout << "printMatrix text: " << text << endl;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            cout << matrix[i][j] << ", ";
        }
        cout << endl;
    }
    return;
}

__global__ void matrixMulShared(float* A, float* B, float* C, int M, int N, int K) {
    // Shared memory for A and B tiles
    __shared__ float s_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float s_B[TILE_WIDTH][TILE_WIDTH];

    // Row and Column index of C element
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    float value = 0.0f;

    // Loop over tiles
    for (int t = 0; t < (K + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {
        if (row < M && t * TILE_WIDTH + threadIdx.x < K)
            s_A[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_WIDTH + threadIdx.x];
        else
            s_A[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < N && t * TILE_WIDTH + threadIdx.y < K)
            s_B[threadIdx.y][threadIdx.x] = B[(t * TILE_WIDTH + threadIdx.y) * N + col];
        else
            s_B[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k)
            value += s_A[threadIdx.y][k] * s_B[k][threadIdx.x];

        __syncthreads();
    }

    if (row < M && col < N)
        C[row * N + col] = value;
}

void cpuMatrixMultiply(float* A, float* B, float* C, int M, int N, int K) {
    for (int row = 0; row < M; ++row)
        for (int col = 0; col < N; ++col) {
            float sum = 0;
            for (int k = 0; k < K; ++k)
                sum += A[row * K + k] * B[k * N + col];
            C[row * N + col] = sum;
        }
}

int main() {
    int M = 256, N = 256, K = 256;

    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    float* h_A = new float[M * K];
    float* h_B = new float[K * N];
    float* h_C = new float[M * N];
    float* h_C_ref = new float[M * N];

    // Initialize matrices
    for (int i = 0; i < M * K; ++i) h_A[i] = static_cast<float>(i % 100) / 100.0f;
    for (int i = 0; i < K * N; ++i) h_B[i] = static_cast<float>((i + 1) % 100) / 100.0f;
    printMatrix("Initialized ", h_A, M, K);
    // CPU reference result
    cpuMatrixMultiply(h_A, h_B, h_C_ref, M, N, K);

    float* d_A, * d_B, * d_C;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid((N + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH);

    matrixMulShared << <dimGrid, dimBlock >> > (d_A, d_B, d_C, M, N, K);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    // Compare results
    bool match = true;
    for (int i = 0; i < M * N; ++i) {
        if (fabs(h_C[i] - h_C_ref[i]) > 1e-3) {
            std::cout << "Mismatch at " << i << ": GPU=" << h_C[i] << ", CPU=" << h_C_ref[i] << "\n";
            match = false;
            break;
        }
    }

    std::cout << (match ? "✅ Matrix multiplication passed.\n" : "❌ Mismatch in results.\n");

    // Cleanup
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_ref;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
