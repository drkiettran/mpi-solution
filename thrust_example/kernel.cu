﻿#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/async/copy.h>
#include <thrust/async/reduce.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/random.h>
#include <numeric>

void example_2() {
	printf("Running example_2\n");
	// Generate random data serially.
	thrust::default_random_engine rng(1337);
	thrust::uniform_real_distribution<double> dist(-50.0, 50.0);
	thrust::host_vector<double> h_vec(32 << 20);
	thrust::generate(h_vec.begin(), h_vec.end(), [&] { return dist(rng); });

	// Transfer to device and compute the sum.
	thrust::device_vector<double> d_vec = h_vec;
	double x = thrust::reduce(d_vec.begin(), d_vec.end(), 0, thrust::plus<int>());
}

void example_1() {
	printf("Running example_1\n");
	// Generate 32M random numbers serially.
	thrust::default_random_engine rng(1337);
	thrust::uniform_int_distribution<int> dist;
	thrust::host_vector<int> h_vec(32 << 20);
	thrust::generate(h_vec.begin(), h_vec.end(), [&] { return dist(rng); });

	// Transfer data to the device.
	thrust::device_vector<int> d_vec = h_vec;

	// Sort data on the device.
	thrust::sort(d_vec.begin(), d_vec.end());

	// Transfer data back to host.
	thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
}

void example_3() {
	printf("Running example_3\n");
	// Generate 32M random numbers serially.
	thrust::default_random_engine rng(123456);
	thrust::uniform_real_distribution<double> dist(-50.0, 50.0);
	thrust::host_vector<double> h_vec(32 << 20);
	thrust::generate(h_vec.begin(), h_vec.end(), [&] { return dist(rng); });

	// Asynchronously transfer to the device.
	thrust::device_vector<double> d_vec(h_vec.size());
	thrust::device_event e = thrust::async::copy(h_vec.begin(), h_vec.end(),
		d_vec.begin());

	// After the transfer completes, asynchronously compute the sum on the device.
	thrust::device_future<double> f0 = thrust::async::reduce(thrust::device.after(e),
		d_vec.begin(), d_vec.end(),
		0.0, thrust::plus<double>());

	// While the sum is being computed on the device, compute the sum serially on
	// the host.
	double f1 = std::accumulate(h_vec.begin(), h_vec.end(), 0.0, thrust::plus<double>());
}

int main() {
	example_1();
	example_2();
	example_3();
}