#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <limits>

const int INF = std::numeric_limits<int>::max();  // Infinity

__global__ void floydWarshallKernel(int* d_dist, int k, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Row index
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Column index

    if (i < n && j < n) {
        int ij = i * n + j;
        int ik = i * n + k;
        int kj = k * n + j;

        // Check if the path i -> k -> j is shorter
        if (d_dist[ik] != INF && d_dist[kj] != INF && d_dist[ik] + d_dist[kj] < d_dist[ij]) {
            d_dist[ij] = d_dist[ik] + d_dist[kj];
        }
    }
}

void floydWarshallCUDA(int* dist, int n) {
    int size = n * n * sizeof(int);
    int* d_dist;

    // Allocate memory on the device
    hipMalloc((void**)&d_dist, size);

    // Copy the distance matrix to the device
    hipMemcpy(d_dist, dist, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 threadsPerBlock(16, 16);  // 16x16 threads per block
    dim3 blocksPerGrid((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Run the Floyd-Warshall algorithm
    for (int k = 0; k < n; ++k) {
        floydWarshallKernel << <blocksPerGrid, threadsPerBlock >> > (d_dist, k, n);
        hipDeviceSynchronize();  // Ensure all threads finish before next iteration
    }

    // Copy the result back to the host
    hipMemcpy(dist, d_dist, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_dist);
}

void printSolution(const int* dist, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (dist[i * n + j] == INF)
                std::cout << "INF ";
            else
                std::cout << dist[i * n + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    // Example graph represented as an adjacency matrix
    int n = 4;
    int graph[16] = {
        0, 3, INF, 7,
        8, 0, 2, INF,
        5, INF, 0, 1,
        2, INF, INF, 0
    };

    // Run Floyd-Warshall algorithm on the GPU
    floydWarshallCUDA(graph, n);

    // Print the solution
    std::cout << "Shortest distances between every pair of vertices:\n";
    printSolution(graph, n);

    return 0;
}
