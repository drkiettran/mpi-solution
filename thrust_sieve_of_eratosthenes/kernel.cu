#include "hip/hip_runtime.h"
﻿#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>

using std::cout;
using std::cin;

#include <iostream>


int main()
{
    int n;
    cout << "Enter the limit: ";
    cin >> n;

    thrust::host_vector<long> tNum(n);
    thrust::sequence(std::begin(tNum), std::end(tNum));
    thrust::transform(std::cbegin(tNum), std::cend(tNum), std::begin(tNum), [](long x)
        {
            bool result = true;
            long stop = (long)std::ceil(std::sqrt((float)x));
            if (x % 2 != 0) {
                for (long i = 3; i < stop; i += 2) {
                    if (x % i == 0) {
                        result = false;
                        break;
                    };
                }
            }
            else {
                result = false;
            }
            if (!result) x = -1;
            return x;
        });
    for (const auto& element : tNum) if (element > 0) std::cout << element << ", ";
    std::cout << std::endl;

    std::cin.ignore();
    return 0;
}