﻿#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

#define size 1024*1024

int main() {
    // Initialize host vectors
    thrust::host_vector<int> h_vec1(size);
    thrust::host_vector<int> h_vec2(size);

    // Fill the vectors with data
    for (int i = 0; i < size; i++) {
        h_vec1[i] = i;
        h_vec2[i] = i * 2;
    }

    // Transfer host vectors to device vectors
    thrust::device_vector<int> d_vec1 = h_vec1;
    thrust::device_vector<int> d_vec2 = h_vec2;
    thrust::device_vector<int> d_result(size);

    // Add vectors element-wise
    thrust::transform(d_vec1.begin(), d_vec1.end(), d_vec2.begin(), d_result.begin(), thrust::plus<int>());

    // Transfer result back to host
    thrust::host_vector<int> h_result = d_result;

    // Display results
    //for (int i = 0; i < size; i++) {
    //    std::cout << h_result[i] << std::endl;
    //}

    return 0;
}
