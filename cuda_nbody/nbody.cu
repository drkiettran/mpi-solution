#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

// ChatGPT generated this code.

// Constants
__device__ const float G = 6.67430e-11f;  // Gravitational constant
const int N = 1000;            // Number of bodies
__device__ const float dt = 0.01f;        // Time step
const int num_timesteps = 100; // Number of simulation steps

// Structure to represent 3D vectors (position, velocity)
struct vec3 {
    float x, y, z;

    __host__ __device__
        vec3() : x(0.0f), y(0.0f), z(0.0f) {}

    __host__ __device__
        vec3(float _x, float _y, float _z) : x(_x), y(_y), z(_z) {}

    // Vector addition
    __host__ __device__
        vec3 operator+(const vec3& b) const {
        return vec3(x + b.x, y + b.y, z + b.z);
    }

    // Scalar multiplication
    __host__ __device__
        vec3 operator*(float scalar) const {
        return vec3(x * scalar, y * scalar, z * scalar);
    }

    // Vector subtraction
    __host__ __device__
        vec3 operator-(const vec3& b) const {
        return vec3(x - b.x, y - b.y, z - b.z);
    }

    // Vector magnitude
    __host__ __device__
        float magnitude() const {
        return sqrtf(x * x + y * y + z * z);
    }
};

// CUDA kernel to compute forces and update velocities and positions
__global__
void nbody_kernel(vec3* positions, vec3* velocities, float* masses, int num_bodies, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= num_bodies) return;

    vec3 pos_i = positions[i];
    vec3 force(0.0f, 0.0f, 0.0f);

    // Calculate net force on body i due to all other bodies
    for (int j = 0; j < num_bodies; ++j) {
        if (i != j) {
            vec3 pos_j = positions[j];
            vec3 diff = pos_j - pos_i;
            float dist = diff.magnitude() + 1e-10f; // Avoid division by zero
            float F = (G * masses[i] * masses[j]) / (dist * dist * dist);
            force = force + diff * F;
        }
    }

    // Update velocity
    vec3 vel_i = velocities[i];
    vec3 accel = force * (1.0f / masses[i]);
    vel_i = vel_i + accel * dt;

    // Update position
    vec3 pos_new = pos_i + vel_i * dt;

    // Write updated position and velocity back
    velocities[i] = vel_i;
    positions[i] = pos_new;
}

int main() {
    // Create random positions, velocities, and masses
    vec3* h_positions = new vec3[N];
    vec3* h_velocities = new vec3[N];
    float* h_masses = new float[N];

    // Initialize random positions, velocities, and masses
    for (int i = 0; i < N; ++i) {
        h_positions[i] = vec3(rand() % 100, rand() % 100, rand() % 100);
        h_velocities[i] = vec3(rand() % 10, rand() % 10, rand() % 10);
        h_masses[i] = rand() % 100 + 1; // Avoid zero mass
    }

    // Allocate device memory
    vec3* d_positions;
    vec3* d_velocities;
    float* d_masses;
    hipMalloc(&d_positions, N * sizeof(vec3));
    hipMalloc(&d_velocities, N * sizeof(vec3));
    hipMalloc(&d_masses, N * sizeof(float));

    // Copy data to device
    hipMemcpy(d_positions, h_positions, N * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_velocities, h_velocities, N * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_masses, h_masses, N * sizeof(float), hipMemcpyHostToDevice);

    // Define block and grid sizes
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    // Run the simulation
    for (int t = 0; t < num_timesteps; ++t) {
        // Launch kernel
        nbody_kernel << <numBlocks, blockSize >> > (d_positions, d_velocities, d_masses, N, dt);
        hipDeviceSynchronize();
    }

    // Copy data back to host
    hipMemcpy(h_positions, d_positions, N * sizeof(vec3), hipMemcpyDeviceToHost);
    hipMemcpy(h_velocities, d_velocities, N * sizeof(vec3), hipMemcpyDeviceToHost);

    // Print the final positions of a few bodies
    std::cout << "Final positions of some bodies:" << std::endl;
    for (int i = 0; i < 5; ++i) {
        std::cout << "Body " << i << ": (" << h_positions[i].x << ", " << h_positions[i].y << ", " << h_positions[i].z << ")" << std::endl;
    }

    // Free memory
    delete[] h_positions;
    delete[] h_velocities;
    delete[] h_masses;
    hipFree(d_positions);
    hipFree(d_velocities);
    hipFree(d_masses);

    return 0;
}
