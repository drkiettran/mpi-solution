#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

/*
* Algorithm: Search for all prime numbers within a range of integer values (i.e., 1-100)
* A prime number is a number that can be divided by 1 and by itself only. Prime number can only
* be an `odd` number.
*
* 1. Create a list of natural number 2,3,4, ... (i.e., 100). None of which is marked.
* 2. Set k=2 the first unmarked number on the list.
* 3. Repeat
*    - Mark all multiples of k between k^2 and n.
*    - Find the smallest number greater than k that is unmarked. Set to this new value.
*    - Until k^2 > n .
*
* The unmarked numbers are primes.
*
* example: prime numbers between 0 and 10.
* 1. [F, F, F, F, F, F, F, F, F]
* 2. k = 2
* 3. Repeat:
*    - k^2 = 4. Multiple of 2 is 4, 6, 8, 10 --> [F, F, T, F, T, F, T, F, T];
*    - Smallest greater than k=2 that is unmarked --> k=3
*    - k^2=3^2=9 > 10? NO
*    - k^2 = 9. Multiple of 3 is 6, 9 --> [F, F, T, F, T, F, T, T, T];
*    - Smallest greater than k=3 that is unmarked --> k=5
*    - k^2=5^2=25 > 10? YES.
* 4. WE ARE DONE. [T, T, T, F, T, F, T, T, T] --> 2, 3, 5, 7
*
*/

__global__ void sieveKernel(bool* d_prime, int n, int sqrt_n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 2;

    if (idx <= sqrt_n && d_prime[idx]) {
        // Mark all multiples of idx as false
        for (int i = idx * idx; i <= n; i += idx) {
            d_prime[i] = false;
        }
    }
}

void sieveOfEratosthenes(int n) {
    // Allocate memory on host (CPU)
    // vector<bool> prime(n + 1, true); 

    // Allocate memory on device (GPU)
    bool *d_prime, *h_prime;
    h_prime = (bool*) malloc(sizeof(bool) * (n + 1));
    for (int i = 0; i < (n + 1); i++) {
        h_prime[i] = true;
    }
    hipMalloc((void**)&d_prime, (n + 1) * sizeof(bool));

    // Copy the data from host to device
    hipMemcpy(d_prime, h_prime, (n + 1) * sizeof(bool), hipMemcpyHostToDevice);

    // Calculate the block size and grid size
    int sqrt_n = sqrt(n);
    int blockSize = 256;
    int gridSize = (sqrt_n + blockSize - 1) / blockSize;

    // Launch the kernel
    sieveKernel << <gridSize, blockSize >> > (d_prime, n, sqrt_n);

    // Copy the result back to the host
    hipMemcpy(h_prime, d_prime, (n + 1) * sizeof(bool), hipMemcpyDeviceToHost);

    // Free the device memory
    hipFree(d_prime);

    // Print the prime numbers
    cout << "Prime numbers up to " << n << " are: ";
    for (int i = 2; i <= n; i++) {
        if (h_prime[i])
            cout << i << " ";
    }
    cout << endl;
}

int main() {
    int n;
    cout << "Enter the limit: ";
    cin >> n;

    sieveOfEratosthenes(n);

    return 0;
}
