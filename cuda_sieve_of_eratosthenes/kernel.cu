#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void sieveKernel(bool* d_prime, int n, int sqrt_n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 2;

    if (idx <= sqrt_n && d_prime[idx]) {
        // Mark all multiples of idx as false
        for (int i = idx * idx; i <= n; i += idx) {
            d_prime[i] = false;
        }
    }
}

void sieveOfEratosthenes(int n) {
    // Allocate memory on host (CPU)
    // vector<bool> prime(n + 1, true); 

    // Allocate memory on device (GPU)
    bool *d_prime, *h_prime;
    h_prime = (bool*) malloc(sizeof(bool) * (n + 1));
    for (int i = 0; i < (n + 1); i++) {
        h_prime[i] = true;
    }
    hipMalloc((void**)&d_prime, (n + 1) * sizeof(bool));

    // Copy the data from host to device
    hipMemcpy(d_prime, h_prime, (n + 1) * sizeof(bool), hipMemcpyHostToDevice);

    // Calculate the block size and grid size
    int sqrt_n = sqrt(n);
    int blockSize = 256;
    int gridSize = (sqrt_n + blockSize - 1) / blockSize;

    // Launch the kernel
    sieveKernel << <gridSize, blockSize >> > (d_prime, n, sqrt_n);

    // Copy the result back to the host
    hipMemcpy(h_prime, d_prime, (n + 1) * sizeof(bool), hipMemcpyDeviceToHost);

    // Free the device memory
    hipFree(d_prime);

    // Print the prime numbers
    cout << "Prime numbers up to " << n << " are: ";
    for (int i = 2; i <= n; i++) {
        if (h_prime[i])
            cout << i << " ";
    }
    cout << endl;
}

int main() {
    int n;
    cout << "Enter the limit: ";
    cin >> n;

    sieveOfEratosthenes(n);

    return 0;
}
