#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>

const int INF = 1e9;  // Use a large value for infinity

// Functor to update the distance matrix in parallel
struct floyd_warshall_functor {
    int* d_dist;
    int n;
    int k;

    floyd_warshall_functor(int* _dist, int _n, int _k)
        : d_dist(_dist), n(_n), k(_k) {}

    __host__ __device__
        void operator()(int idx) {
        int i = idx / n;
        int j = idx % n;

        int ik = i * n + k;
        int kj = k * n + j;
        int ij = i * n + j;

        if (d_dist[ik] != INF && d_dist[kj] != INF) {
            d_dist[ij] = min(d_dist[ij], d_dist[ik] + d_dist[kj]);
        }
    }
};

void floydWarshallThrust(int* dist, int n) {
    // Move the distance matrix to the GPU
    thrust::device_vector<int> d_dist(dist, dist + n * n);

    for (int k = 0; k < n; ++k) {
        // Use thrust::for_each to update all pairs (i, j)
        thrust::for_each(thrust::device,
            thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(n * n),
            floyd_warshall_functor(thrust::raw_pointer_cast(d_dist.data()), n, k));
    }

    // Copy the result back to the host
    thrust::copy(d_dist.begin(), d_dist.end(), dist);
}

void printSolution(const int* dist, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (dist[i * n + j] == INF)
                std::cout << "INF ";
            else
                std::cout << dist[i * n + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    // Example graph represented as an adjacency matrix
    int n = 4;
    int graph[16] = {
        0, 3, INF, 7,
        8, 0, 2, INF,
        5, INF, 0, 1,
        2, INF, INF, 0
    };

    // Run Floyd-Warshall algorithm using Thrust
    floydWarshallThrust(graph, n);

    // Print the solution
    std::cout << "Shortest distances between every pair of vertices:\n";
    printSolution(graph, n);

    return 0;
}
